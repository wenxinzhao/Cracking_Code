#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>



#define DATA_SIZE   1048576
#define BLOCK_NUM   32
#define THREAD_NUM   256

int data[DATA_SIZE];
clock_t  clockBegin, clockEnd; 

__global__ static void sumOfSquares(int *num, int* result,
    clock_t* time)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    int sum = 0;
    int i;
    if(tid == 0) time[bid] = clock();
    for(i = bid * THREAD_NUM + tid; i < DATA_SIZE;
        i += BLOCK_NUM * THREAD_NUM) {
       sum += num[i] * num[i];
    }

    result[bid * THREAD_NUM + tid] = sum;
    if(tid == 0) time[bid + BLOCK_NUM] = clock();
}

void PrintfContainerElapseTime(char *pszContainerName, char *pszOperator, long lElapsetime)
{
	printf("%s %s time %dsec\n", pszContainerName, pszOperator, lElapsetime);
}

void GenerateNumbers(int *number, int size)
{
    for(int i = 0; i < size; i++) {
        number[i] = rand() % 10;
    }
}

bool InitCUDA()
{
    int count;

    hipGetDeviceCount(&count);
    if(count == 0) {
        fprintf(stderr, "There is no device./n");
        return false;
    }

    int i;
    for(i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if(prop.major >= 1) {
                break;
            }
        }
    }

    if(i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x./n");
        return false;
    }

    hipSetDevice(i);

    return true;
}

int main()
{
    if(!InitCUDA()) {
        return 0;
    }

    printf("CUDA initialized.\n");

	GenerateNumbers(data, DATA_SIZE);

    /*int* gpudata, *result;
    cudaMalloc((void**) &gpudata, sizeof(int) * DATA_SIZE);
    cudaMalloc((void**) &result, sizeof(int));
    cudaMemcpy(gpudata, data, sizeof(int) * DATA_SIZE,
        cudaMemcpyHostToDevice);

	sumOfSquares<<<1, 1, 0>>>(gpudata, result);

    int sum;
    cudaMemcpy(&sum, result, sizeof(int), cudaMemcpyDeviceToHost);
    cudaFree(gpudata);
    cudaFree(result);

    printf("sum: %d\n", sum);*/

	int* gpudata, *result;
    clock_t* time;
    hipMalloc((void**) &gpudata, sizeof(int) * DATA_SIZE);
    hipMalloc((void**) &result,
        sizeof(int) * THREAD_NUM * BLOCK_NUM);
    hipMalloc((void**) &time, sizeof(clock_t) * BLOCK_NUM * 2);
    hipMemcpy(gpudata, data, sizeof(int) * DATA_SIZE,
        hipMemcpyHostToDevice);

    sumOfSquares<<<BLOCK_NUM, THREAD_NUM, 0>>>(gpudata, result, 
        time);

    int sum[THREAD_NUM * BLOCK_NUM];
    clock_t time_used[BLOCK_NUM * 2];
    hipMemcpy(&sum, result, sizeof(int) * THREAD_NUM * BLOCK_NUM,
        hipMemcpyDeviceToHost);
    hipMemcpy(&time_used, time, sizeof(clock_t) * BLOCK_NUM * 2,
        hipMemcpyDeviceToHost);
    hipFree(gpudata);
    hipFree(result);
    hipFree(time);

    int final_sum = 0;
    for(int i = 0; i < THREAD_NUM * BLOCK_NUM; i++) {
        final_sum += sum[i];
    }

    clock_t min_start, max_end;
    min_start = time_used[0];
    max_end = time_used[BLOCK_NUM];
    for(int i = 1; i < BLOCK_NUM; i++) {
        if(min_start > time_used[i])
            min_start = time_used[i];
        if(max_end < time_used[i + BLOCK_NUM])
            max_end = time_used[i + BLOCK_NUM];
    }

    printf("sum: %d  time: %d\n", final_sum, max_end - min_start);
	



	final_sum = 0;
    for(int i = 0; i < THREAD_NUM; i++) {
        final_sum += sum[i];
    }

    printf("sum: %d  time: %d\n", final_sum, time_used);

    final_sum = 0;
    for(int i = 0; i < DATA_SIZE; i++) {
      final_sum += data[i] * data[i];
    }
    printf("sum (CPU): %d\n", final_sum);


    getchar();
    return 0;
}

